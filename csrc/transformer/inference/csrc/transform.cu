#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#ifndef __HIP_PLATFORM_HCC__
#include <hip/hip_runtime_api.h>
#endif
#include "conversion_utils.h"
#include "inference_cuda_layers.h"
namespace cg = cooperative_groups;

// only used to avoid compilation error due to lack of definition.
#ifndef BF16_AVAILABLE
using __hip_bfloat162 = __half2;
#endif

// Bias add

__global__ void bias_add_transform_0213(float* output,
                                        float* k_cache,
                                        float* v_cache,
                                        const float* vals,
                                        const float* bias,
                                        int hidden_dim,
                                        int seq_length,
                                        unsigned seq_offset,
                                        int heads,
                                        int rotary_dim,
                                        bool rotate_half,
                                        bool rotate_every_two,
                                        int head_ext,
                                        int max_out_tokens)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    int d2_out_stride = d2_stride * (cnt == 0 ? seq_length : max_out_tokens);
    int d0_out_stride = hidden_dim * (cnt == 0 ? seq_length : max_out_tokens);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* output_vec =
        reinterpret_cast<float4*>(cnt == 0 ? output : (cnt == 1 ? k_cache : v_cache));

    vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    output_vec += (d1 * d2_stride);
    output_vec += (d0 * d0_out_stride);
    output_vec += (d2 * d2_out_stride);

    unsigned seq_id = d1 + seq_offset;
    float4 inputs = vals_vec[d3];
    int lane = d3 & 0x1f;
    if (cnt < 2 && rotary_dim > 0 && d3 < rotary_dim) {
        float4 q = vals_vec[d3];
        float2* q_f = reinterpret_cast<float2*>(&q);
        if (rotate_every_two) {
#pragma unroll
            for (int o = 0; o < 2; o++) {
                float inv_freq = (float)(((d3 << 1) + o) * 2) / (float)(rotary_dim << 2);
                inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
                q_f[o].x = (-1.0 * q_f[o].y * sinf(inv_freq) + q_f[o].x * cosf(inv_freq));
                q_f[o].y = (q_f[o].x * sinf(inv_freq) + q_f[o].y * cosf(inv_freq));
            }
        }
        output_vec[d3] = q;
    } else
        output_vec[d3] = inputs;
}

#define ATTN_H 3
#define MAX_SEQ_LINE 10

template <typename T>
__global__ void bias_add_transform_0213(T* output,  // q
                                        T* k_cache,
                                        T* v_cache,
                                        const T* vals,  // qkv
                                        const T* bias,
                                        int hidden_dim,
                                        int seq_length,
                                        unsigned seq_offset,
                                        int all_tokens,
                                        int heads,
                                        int rotary_dim,
                                        bool rotate_half,
                                        bool rotate_every_two,
                                        int head_ext,
                                        int max_out_tokens)
{
    using T2 =
        typename std::conditional<std::is_same<T, __half>::value, __half2, __hip_bfloat162>::type;
    unsigned half_dim = (rotary_dim << 3) >> 1;
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    int d2_out_stride = d2_stride * (cnt == 0 ? seq_length : max_out_tokens);
    int d0_out_stride = hidden_dim * (cnt == 0 ? seq_length : max_out_tokens);

    float4 vals_arr;
    float4 output_arr;

    T2* vals_half = reinterpret_cast<T2*>(&vals_arr);
    T2* output_half = reinterpret_cast<T2*>(&output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* output_vec =
        reinterpret_cast<float4*>(cnt == 0 ? output : (cnt == 1 ? k_cache : v_cache));

    vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    output_vec += (d1 * d2_stride);
    output_vec += (d0 * d0_out_stride);
    output_vec += (d2 * d2_out_stride);

    unsigned seq_id = d1 + seq_offset;

    int lane = d3 & 0x1f;
    if (cnt < 2 && rotary_dim > 0 && d3 < rotary_dim) {
        float4 q = vals_vec[d3];
        T2* q_h = reinterpret_cast<T2*>(&q);
        if (rotate_every_two) {
#pragma unroll
            for (int o = 0; o < 4; o++) {
                float inv_freq = (float)(((d3 << 2) + o) * 2) / (float)(rotary_dim << 3);
                inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
                float q_data[2];
                q_data[0] = conversion::to<float>(q_h[o].x);
                q_data[1] = conversion::to<float>(q_h[o].y);
                q_h[o].x = conversion::to<T>(-1.0 * q_data[1] * sinf(inv_freq) +
                                             q_data[0] * cosf(inv_freq));
                q_h[o].y =
                    conversion::to<T>(q_data[0] * sinf(inv_freq) + q_data[1] * cosf(inv_freq));
            }
        }
        output_vec[d3] = q;
    } else
        output_vec[d3] = vals_vec[d3];
}

// [B S C*H] - > C * [B A S N]
template <>
void launch_bias_add_transform_0213<float>(float* output,
                                           float* k_cache,
                                           float* v_cache,
                                           const float* vals,
                                           const float* bias,
                                           int batch_size,
                                           int seq_length,
                                           unsigned seq_offset,
                                           int all_tokens,
                                           int hidden_dim,
                                           int heads,
                                           int rotary_dim,
                                           bool rotate_half,
                                           bool rotate_every_two,
                                           hipStream_t stream,
                                           int trans_count,
                                           int max_out_tokens)
{
    hidden_dim >>= 2;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;

    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));

    bias_add_transform_0213<<<grid_dim, block_dim, 0, stream>>>(output,
                                                                k_cache,
                                                                v_cache,
                                                                vals,
                                                                bias,
                                                                hidden_dim,
                                                                seq_length,
                                                                seq_offset,
                                                                heads,
                                                                rotary_dim >> 2,
                                                                rotate_half,
                                                                rotate_every_two,
                                                                head_ext,
                                                                max_out_tokens);
}

template <typename T>
void launch_bias_add_transform_0213(T* output,
                                    T* k_cache,
                                    T* v_cache,
                                    const T* vals,
                                    const T* bias,
                                    int batch_size,
                                    int seq_length,
                                    unsigned seq_offset,
                                    int all_tokens,
                                    int hidden_dim,
                                    int heads,
                                    int rotary_dim,
                                    bool rotate_half,
                                    bool rotate_every_two,
                                    hipStream_t stream,
                                    int trans_count,
                                    int max_out_tokens)
{
    hidden_dim >>= 3;
    int head_ext = 1;  // (hidden_dim - 1) / MAX_THREADS + 1;
    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));
    bias_add_transform_0213<<<grid_dim, block_dim, 0, stream>>>(output,
                                                                k_cache,
                                                                v_cache,
                                                                vals,
                                                                bias,
                                                                hidden_dim,
                                                                seq_length,
                                                                seq_offset,
                                                                all_tokens,
                                                                heads,
                                                                rotary_dim >> 3,
                                                                rotate_half,
                                                                rotate_every_two,
                                                                head_ext,
                                                                max_out_tokens);
}

#define INSTANTIATE_LAUNCH_BIAS_ADD_TRANSFORM_0213(T)             \
    template void launch_bias_add_transform_0213<T>(T*,           \
                                                    T*,           \
                                                    T*,           \
                                                    const T*,     \
                                                    const T*,     \
                                                    int,          \
                                                    int,          \
                                                    unsigned,     \
                                                    int,          \
                                                    int,          \
                                                    int,          \
                                                    int,          \
                                                    bool,         \
                                                    bool,         \
                                                    hipStream_t, \
                                                    int,          \
                                                    int)

#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_BIAS_ADD_TRANSFORM_0213(__hip_bfloat16);
#endif
INSTANTIATE_LAUNCH_BIAS_ADD_TRANSFORM_0213(__half);

// Bias add

__global__ void pad_add_transform_0213(float* output,
                                       const float* vals,
                                       int hidden_dim,
                                       int seq_length,
                                       int padded_seq_len,
                                       int heads,
                                       int padded_head_size)
{
}

template <typename T>
__global__ void pad_add_transform_0213(T* output,
                                       const T* vals,
                                       int hidden_dim,
                                       int seq_length,
                                       int padded_seq_len,
                                       int heads,
                                       int padded_head_size)
{
    using T2 =
        typename std::conditional<std::is_same<T, __half>::value, __half2, __hip_bfloat162>::type;
    float4 ZERO;
    const T2 zero_h = conversion::to<T2>(0.f);
    T2* ZERO_h = reinterpret_cast<T2*>(&ZERO);
#pragma unroll
    for (int i = 0; i < 4; i++) ZERO_h[i] = zero_h;

    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;                             // Batch
    int d1 = blockIdx.y * blockDim.z + threadIdx.z;  // Sequence ID (0-127)
    int d2 = threadIdx.y;                            // Head (0-11)
    int d3 = threadIdx.x;                            // Values (groups of 4)

    int d2_out_stride = padded_head_size * padded_seq_len;
    int d0_out_stride = heads * d2_out_stride;

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* output_vec = reinterpret_cast<float4*>(output);

    vals_vec += (d0 * d0_stride);
    vals_vec += (d1 * d1_stride);
    vals_vec += (d2 * d2_stride);

    output_vec += (d1 * padded_head_size);
    output_vec += (d0 * d0_out_stride);
    output_vec += (d2 * d2_out_stride);

    if (d3 < d2_stride && d1 < seq_length)
        output_vec[d3] = vals_vec[d3];
    else
        output_vec[d3] = ZERO;
}

// [B S C*H] - > C * [B A S N]
template <>
void launch_pad_add_transform_0213<float>(float* output,
                                          const float* vals,
                                          int batch_size,
                                          int hidden_dim,
                                          int seq_length,
                                          int padded_seq_len,
                                          int heads,
                                          int padded_head_size,
                                          hipStream_t stream)
{
}

template <typename T>
void launch_pad_add_transform_0213(T* output,
                                   const T* vals,
                                   int batch_size,
                                   int hidden_dim,
                                   int seq_length,
                                   int padded_seq_len,
                                   int heads,
                                   int padded_head_size,
                                   hipStream_t stream)
{
    hidden_dim >>= 3;
    dim3 block_dim((padded_head_size >> 3), heads, 2);
    dim3 grid_dim(batch_size, padded_seq_len / 2);
    pad_add_transform_0213<<<grid_dim, block_dim, 0, stream>>>(
        output, vals, hidden_dim, seq_length, padded_seq_len, heads, padded_head_size >> 3);
}

#define INSTANTIATE_LAUNCH_PAD_ADD_TRANSFORM_0213_SIMPLE(T) \
    template void launch_pad_add_transform_0213<T>(         \
        T*, const T*, int, int, int, int, int, int, hipStream_t);

INSTANTIATE_LAUNCH_PAD_ADD_TRANSFORM_0213_SIMPLE(__half);
#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_PAD_ADD_TRANSFORM_0213_SIMPLE(__hip_bfloat16);
#endif

// Bias add
template <typename T>
__global__ void bias_add_transform_0213(T* output,
                                        const T* vals,
                                        const T* bias,
                                        int hidden_dim,
                                        int seq_length,
                                        int heads,
                                        int head_ext);

template <>
__global__ void bias_add_transform_0213<float>(float* output,
                                               const float* vals,
                                               const float* bias,
                                               int hidden_dim,
                                               int seq_length,
                                               int heads,
                                               int head_ext)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    float4 inputs = vals_vec[d0 * d0_stride * (gridDim.z / head_ext) + cnt * d1_stride +
                             d1 * d1_stride * (gridDim.z / head_ext) + d2 * d2_stride + d3];
    float4 biases = bias_vec[cnt * d1_stride + d2 * d2_stride + d3];

    float4 outputs;
    outputs.x = inputs.x + biases.x;
    outputs.y = inputs.y + biases.y;
    outputs.z = inputs.z + biases.z;
    outputs.w = inputs.w + biases.w;

    output_vec[cnt * d0_out_stride * gridDim.x + d0 * d0_out_stride + d1 * d1_out_stride +
               d2 * d2_out_stride + d3] = outputs;
}

template <typename T>
__global__ void bias_add_transform_0213(T* output,
                                        const T* vals,
                                        const T* bias,
                                        int hidden_dim,
                                        int seq_length,
                                        int heads,
                                        int head_ext)
{
    using T2 =
        typename std::conditional<std::is_same<T, __half>::value, __half2, __hip_bfloat162>::type;
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    float4 vals_arr;
    float4 bias_arr;
    float4 output_arr;
    T2* vals_half = reinterpret_cast<T2*>(&vals_arr);
    T2* bias_half = reinterpret_cast<T2*>(&bias_arr);
    T2* output_half = reinterpret_cast<T2*>(&output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    bias_vec += (cnt * d1_stride);
    bias_vec += (d2 * d2_stride);

    output_vec += (cnt * d0_stride * gridDim.x);
    output_vec += (d1 * d2_stride);
    output_vec += (d0 * d0_stride);
    output_vec += (d2 * d2_out_stride);

    bias_arr = bias_vec[d3];
    vals_arr = vals_vec[d3];

    output_half[0] = vals_half[0] + bias_half[0];
    output_half[1] = vals_half[1] + bias_half[1];
    output_half[2] = vals_half[2] + bias_half[2];
    output_half[3] = vals_half[3] + bias_half[3];
    output_vec[d3] = output_arr;
}

template <typename T>
__global__ void bias_add_transform_0213_v2(T* output,
                                           const T* vals,
                                           const T* bias,
                                           int hidden_dim,
                                           int seq_length,
                                           int heads)
{
    using T2 =
        typename std::conditional<std::is_same<T, __half>::value, __half2, __hip_bfloat162>::type;
    __shared__ float4 in_data[3072];

    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;
    int iteration_stride = d1_stride * blockDim.z;  // Hidden * 3 / 8
    int batch_stride = d0_stride * blockDim.z;      // Hidden * S * 3 / 8

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;    // Batch
    int d1 = blockIdx.y;    // Sequence ID (0-127)
    int cnt = threadIdx.z;  // blockIdx.z; // Hidden count
    int d2 = threadIdx.y;   // Head (0-11)
    int d3 = threadIdx.x;   // Values (groups of 4)

    float4 vals_arr[1];
    float4 bias_arr[1];
    float4 output_arr[1];
    T2* vals_half = reinterpret_cast<T2*>(vals_arr);
    T2* bias_half = reinterpret_cast<T2*>(bias_arr);
    T2* output_half = reinterpret_cast<T2*>(output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    int iter_index = cnt * d1_stride + d2 * d2_stride + d3;
    int input_offset = d0 * batch_stride + d1 * (iteration_stride << 1);
    bias_arr[0] = bias_vec[iter_index];

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_id = iter * iteration_stride + iter_index;
        vals_arr[0] = vals_vec[input_offset + iter_id];

        output_half[0] = vals_half[0] + bias_half[0];
        output_half[1] = vals_half[1] + bias_half[1];
        output_half[2] = vals_half[2] + bias_half[2];
        output_half[3] = vals_half[3] + bias_half[3];

        in_data[iter_id] = output_arr[0];
    }
    __syncthreads();

    iteration_stride = blockDim.z * (blockDim.y >> 1);
    int matrix_stride = (d0_out_stride * gridDim.x);
    int head_count = (d2 >> 1) + cnt * (blockDim.y >> 1);

    int out_index = d0 * d0_out_stride + d1 * (d1_out_stride << 1) + d3 + (d2 % 2) * d2_stride;

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_row = (iter * iteration_stride) + head_count;
        int iter_offset =
            (iter_row % blockDim.y) * d2_out_stride + (iter_row / blockDim.y) * matrix_stride;
        output_vec[out_index + iter_offset] =
            in_data[iter_row * d2_stride + d3 + (d2 % 2) * (d1_stride * blockDim.z)];
    }
}

template <typename T>
__global__ void transform4d_0213(T* out,
                                 const T* in,
                                 int heads,
                                 int seq_length,
                                 int hidden_dim,
                                 int head_ext);

template <>
__global__ void transform4d_0213<float>(float* out,
                                        const float* in,
                                        int heads,
                                        int seq_length,
                                        int hidden_dim,
                                        int head_ext)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = d0_stride / heads;
    int d2_stride = hidden_dim / heads;

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = hidden_dim;

    int d0 = blockIdx.x;                                        // Batch
    int d1 = blockIdx.y / ((seq_length - 1) / blockDim.y + 1);  // Head
    int d2 = (threadIdx.y + blockDim.y * blockIdx.y) % seq_length;
    int cnt = blockIdx.z;
    int d3 = threadIdx.x;  // Values (groups of 8)

    if (d2 < seq_length) {
        const float4* in_vec = reinterpret_cast<const float4*>(in);
        float4* out_vec = reinterpret_cast<float4*>(out);

        float4 vals_vec = in_vec[cnt * d0_stride * gridDim.x + d0 * d0_stride + d1 * d1_stride +
                                 d2 * d2_stride + d3];
        out_vec[d0 * d0_out_stride * gridDim.z + cnt * d2_out_stride + d1 * d1_out_stride +
                d2 * d2_out_stride * gridDim.z + d3] = vals_vec;
    }
}

template <typename T>
__global__ void transform4d_0213(T* out,
                                 const T* in,
                                 int heads,
                                 int seq_length,
                                 int hidden_dim,
                                 int head_ext)
{
    int d0_stride = hidden_dim * (seq_length / head_ext);
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head
    int d2 = blockIdx.z / head_ext;                                       // Sequence
    int cnt = blockIdx.y;                                                 // Hidden count
    int d3 = threadIdx.x;                                                 // Values (groups of 8)

    const float4* in_vec = reinterpret_cast<const float4*>(in);
    float4* out_vec = reinterpret_cast<float4*>(out);

    in_vec += (cnt * d0_stride * gridDim.x);
    in_vec += (d0 * d0_stride);
    in_vec += (d2 * d2_stride);
    in_vec += (d1 * d2_stride * seq_length);

    out_vec += (cnt * d1_stride);
    out_vec += (d1 * d2_stride);
    out_vec += (d0 * d0_stride * gridDim.y);
    out_vec += (d2 * d1_stride * gridDim.y);

    out_vec[d3] = in_vec[d3];
}

template <typename T>
__global__ void transform4d_0213_v2(T* out, const T* in, int heads, int seq_length, int hidden_dim)
{
    __shared__ float4 in_data[3072];

    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;    // Batch
    int d1 = threadIdx.y;   // Head
    int d2 = blockIdx.y;    // Sequence
    int cnt = threadIdx.z;  // Hidden count
    int d3 = threadIdx.x;   // Values (groups of 8)

    const float4* in_vec = reinterpret_cast<const float4*>(in);
    float4* out_vec = reinterpret_cast<float4*>(out);

    int input_offset = d0 * d0_stride + d2 * (d2_stride << 1) + d3 + (d1 % 2) * d2_stride;
    int head_count = (d1 >> 1) + cnt * (blockDim.y >> 1);
    int iteration_stride = blockDim.z * (blockDim.y >> 1);
    int matrix_stride = (d0_stride * gridDim.x);

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_row = iter * iteration_stride + head_count;
        int iter_offset = (iter_row % blockDim.y) * d2_stride;

        in_data[d3 + iter_offset + (iter_row / blockDim.y + (d1 % 2) * blockDim.z) * d1_stride] =
            in_vec[input_offset + iter_offset * seq_length +
                   (iter_row / blockDim.y) * matrix_stride];
    }
    __syncthreads();

    iteration_stride = d1_stride * blockDim.z;
    int iter_index = cnt * d1_stride + d1 * d2_stride + d3;
    int output_offset = d0 * d0_stride * blockDim.z + d2 * (iteration_stride << 1);

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_id = iter * iteration_stride + iter_index;
        out_vec[output_offset + iter_id] = in_data[iter_id];
    }
}

// 3 * [B A S N] - > [B S C*H]
template <>
void launch_transform4d_0213<float>(float* out,
                                    const float* in,
                                    int batch_size,
                                    int heads,
                                    int seq_length,
                                    int hidden_dim,
                                    hipStream_t stream,
                                    int trans_count)
{
    hidden_dim >>= 2;
    dim3 grid_dims(batch_size, heads * ((seq_length - 1) / 8 + 1), trans_count);
    dim3 block_dims(hidden_dim / heads, 8);
    transform4d_0213<float>
        <<<grid_dims, block_dims, 0, stream>>>(out, in, heads, seq_length, hidden_dim, 1);
}

template <typename T>
void launch_transform4d_0213<T>(T* out,
                                const T* in,
                                int batch_size,
                                int heads,
                                int seq_length,
                                int hidden_dim,
                                hipStream_t stream,
                                int trans_count)
{
    hidden_dim >>= 3;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;
    dim3 grid_dims(batch_size, trans_count, (seq_length * head_ext));
    dim3 block_dims(hidden_dim / heads, (heads / head_ext));
    transform4d_0213<<<grid_dims, block_dims, 0, stream>>>(
        out, in, heads, seq_length, hidden_dim, head_ext);
}

#define INSTANTIATE_2B_LAUNCH_TRANSFORM4D(T) \
    template void launch_transform4d_0213<T>(T*, const T*, int, int, int, int, hipStream_t, int);

INSTANTIATE_2B_LAUNCH_TRANSFORM4D(__half)
#ifdef BF16_AVAILABLE
INSTANTIATE_2B_LAUNCH_TRANSFORM4D(__hip_bfloat16)
#endif

__global__ void transform_multi_query(float* query,
                                      float* key,
                                      float* value,
                                      float* k_cache,
                                      float* v_cache,
                                      const float* vals,
                                      int hidden_dim,
                                      int seq_length,
                                      int all_tokens,
                                      int heads,
                                      int max_out_tokens)
{
}

#define ATTN_H 3
#define MAX_SEQ_LINE 10

template <typename T>
__global__ void transform_multi_query(T* query,
                                      T* key,
                                      T* value,
                                      T* k_cache,
                                      T* v_cache,
                                      const T* vals,  // qkv
                                      int seq_length,
                                      int all_tokens,
                                      int heads,
                                      int num_kv,
                                      int blks,
                                      int query_heads,
                                      int fused_dim,
                                      int qkv_dim,
                                      int hidden_dim,
                                      int max_out_tokens)
{
    using T2 =
        typename std::conditional<std::is_same<T, __half>::value, __half2, __hip_bfloat162>::type;
    int d0_stride = fused_dim * seq_length;
    int d1_stride = fused_dim;
    int d2_stride = blockDim.x;

    int d0 = blockIdx.x;   // Batch
    int d1 = blockIdx.y;   // Sequence ID (0-127)
    int cnt = 0;//blockIdx.z / blks;  // kv count
    int blk_count = blockIdx.z ;//% blks;
    int d2 = threadIdx.y + blk_count * blockDim.y;
    int d3 = threadIdx.x;  // Values (groups of 4)

    int d2_out_stride = d2_stride * seq_length;
    int d0_out_stride = hidden_dim * seq_length;

    int d2_out_stride_kv = d2_stride * all_tokens;
    int d0_out_stride_kv = hidden_dim * all_tokens;

    float4 vals_arr;
    float4 output_arr;

    T2* vals_half = reinterpret_cast<T2*>(&vals_arr);
    T2* output_half = reinterpret_cast<T2*>(&output_arr);
    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* cache = reinterpret_cast<float4*>(d2 < (query_heads << 1) ? k_cache : v_cache);
    float4* output_vec = reinterpret_cast<float4*>(
        d2 < query_heads ? query : (d2 < (query_heads << 1) ? key : value));

    vals_vec += (d0 * d0_stride);
    vals_vec += (d1 * d1_stride);
    vals_vec += (cnt * qkv_dim);

    if (d2 < query_heads) {
        vals_vec += (d2 * d2_stride);
    } else {
        if (d2 < (query_heads << 1))
            vals_vec += (query_heads * d2_stride);
        else
            vals_vec += ((query_heads + 1) * d2_stride);
    }

    output_vec += (d1 * d2_stride);
    output_vec += (d0 * (d2 < query_heads ? d0_out_stride : d0_out_stride_kv));
    output_vec += (((d2 % query_heads)) *
                   (d2 < query_heads ? d2_out_stride : d2_out_stride_kv));

    if (d1 < seq_length || d2 >= query_heads) {
        if (d2 < query_heads)
            output_vec[d3] = vals_vec[d3];
        else {
            if (d1 < seq_length) {
                output_vec += (all_tokens - seq_length) * d2_stride;
                output_vec[d3] = vals_vec[d3];
            } 
            else {
                output_vec -= d2_stride;
                cache += (d2_stride * max_out_tokens * (d2 < (query_heads << 1) ? heads : num_kv)) * d0 + 
                         d2_stride * (d1 - (seq_length == 1 ? 1 : 0)) +
                         (d2 < (query_heads << 1) ? (d2 % query_heads) : cnt) * (d2_stride * max_out_tokens);
                output_vec[d3] = cache[d3];
            }
        }
    }

    if (((d1 < seq_length && seq_length > 1) || d1 == 0) && (d2 >= query_heads && d2 <= (query_heads << 1))) {
        cache += (d2_stride * max_out_tokens * ((d2 < (query_heads << 1)) ? heads : num_kv)) * d0 +
                 d2_stride * (all_tokens - seq_length + d1) + 
                 (d2 < (query_heads << 1) ? (d2 % query_heads) : cnt) * (d2_stride * max_out_tokens);
        cache[d3] = vals_vec[d3];
    }
}

// [B S C*H] - > C * [B A S N]
template <>
void launch_transform_multi_query<float>(float* query,
                                         float* key,
                                         float* value,
                                         float* k_cache,
                                         float* v_cache,
                                         const float* vals,
                                         int batch_size,
                                         int seq_length,
                                         int all_tokens,
                                         int hidden_dim,
                                         int heads,
                                         int num_kv,
                                         hipStream_t stream,
                                         int max_out_tokens)
{
}
template <typename T>
void launch_transform_multi_query(T* query,
                                  T* key,
                                  T* value,
                                  T* k_cache,
                                  T* v_cache,
                                  const T* vals,
                                  int batch_size,
                                  int seq_length,
                                  int all_tokens,
                                  int hidden_dim,
                                  int heads,
                                  int num_kv,
                                  hipStream_t stream,
                                  int max_out_tokens)
{
    hidden_dim >>= 3;
    int max_thread_blk = 1024 / (hidden_dim / heads);
    int threadblks = (heads / num_kv) * 3;
    int launch_blks = threadblks;
    if (launch_blks > max_thread_blk) launch_blks = max_thread_blk;
    int num_blks = (threadblks - 1) / launch_blks + 1;
    dim3 block_dim(hidden_dim / heads, (heads / num_kv) * 3);
    dim3 grid_dim(batch_size, all_tokens, 1);
    transform_multi_query<<<grid_dim, block_dim, 0, stream>>>(
        query,
        key,
        value,
        k_cache,
        v_cache,
        vals,
        seq_length,
        all_tokens,
        heads,
        num_kv,
        num_blks,
        heads / num_kv,
        hidden_dim + num_kv * 2 * (hidden_dim / heads),
        (heads / num_kv + 2) * (hidden_dim / heads),
        hidden_dim,
        max_out_tokens);
}

#define INSTANTIATE_LAUNCH_TRANSFORM_MULTI_QUERY(T) \
    template void launch_transform_multi_query<T>(  \
        T*, T*, T*, T*, T*, const T*, int, int, int, int, int, int, hipStream_t, int)

#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_TRANSFORM_MULTI_QUERY(__hip_bfloat16);
#endif
INSTANTIATE_LAUNCH_TRANSFORM_MULTI_QUERY(__half);
